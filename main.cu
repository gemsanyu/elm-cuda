#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdio.h>
#include <stdlib.h>

#include "local_lib.h"

#define M 5
#define K 4
#define IDX2C(i,j,ld) (((i)*(ld))+(j))

int main(int argc, char ** argv){
  hipError_t cudaStat; // hipMalloc hipblasStatus_t
  hipblasStatus_t cublasStat; // CUBLAS functions hipblasStatus_t
  hipblasHandle_t cublasH; // CUBLAS context
  hipsolverHandle_t cusolverH;
  hipsolverStatus_t cusolverStat;

  cublasStat = hipblasCreate(&cublasH); // initialize CUBLAS context
  cusolverStat = hipsolverDnCreate(&cusolverH);
  assert(cusolverStat == HIPSOLVER_STATUS_SUCCESS);

  float *a, *aInv; // mxk matrix a on the host
  a = (float*) malloc (M*K* sizeof(float)); // host memory for a
  aInv = (float*) malloc (M*K* sizeof(float));

  int i,j,ind = 1;
  for(i=0;i<M;i++){ // 11 ,17 ,23 ,29 ,35
    for(j=0;j<K;j++){ // 12 ,18 ,24 ,30 ,36
      a[IDX2C(i,j,K)]=(float)ind++; // 13 ,19 ,25 ,31 ,37
    } // 14 ,20 ,26 ,32 ,38
  } // 15 ,21 ,27 ,33 ,39

  printf ("a:\n");
  for (i=0;i<M;i ++){
    for (j=0;j<K;j ++){
      printf (" %.8f",a[ IDX2C(i,j,K)]);
    }
  printf ("\n");
  }

  float * d_a; // d_a - a on the device
  cudaStat = hipMalloc (( void **)& d_a ,M*K* sizeof (float)); // device
  cublasStat = hipblasSetMatrix(M, K, sizeof(float), a, M, d_a, M); //a -> d_a
  float * d_Ainv = getPseudoInverse(cublasH, cusolverH, d_a, M, K);
  cublasStat = hipblasGetMatrix (K,M, sizeof (float) ,d_Ainv , K, aInv, K); // cp d_c - >c

  printf ("a:\n");
  for (i=0;i<K;i ++){
    for (j=0;j<M;j ++){
      printf (" %.8f",aInv[ IDX2C(i,j,M)]);
    }
  printf ("\n");
  }

  cublasStat = hipblasDestroy(cublasH);
  cusolverStat = hipsolverDnDestroy(cusolverH);

}
