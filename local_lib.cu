#include "hip/hip_runtime.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <math.h>
#include <stdio.h>

#include "local_lib.h"

#define BLOCK_SIZE 256
#define GRID_SIZE 50000
#define EPS 10e-5

hipblasStatus_t matMul(hipblasHandle_t cublasH, float *d_A, float *d_B, float *d_result,
  int m, int k, int n, bool isRowMajor, float alfa, float beta){
  hipblasStatus_t stat;
  if (isRowMajor){
    stat=hipblasSgemm(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,n,m,k,&alfa,
      d_B,n,d_A,k,&beta,d_result,n);
  } else {
    stat=hipblasSgemm(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&alfa,
      d_A,m,d_B,k,&beta,d_result,m);
  }
  return stat;
}

__global__ void d_ActivationFunction(float *d_A, int *d_m, int *d_n){
  int m = (*d_m);
  int n = (*d_n);
  int size = m*n;
  int bIdx = blockIdx.x;
  int tIdx = threadIdx.x;
  int stride = blockDim.x;
  int id = bIdx*stride + tIdx;
  for(int i=id;i<size;i+=stride){
    d_A[i] = 1.0 / (1.0 + exp(-d_A[i]));
  }
}

void activationFunction(float *d_A, int m, int n){
  int gridSize = (m*n/BLOCK_SIZE + 1);
  gridSize = min(gridSize, GRID_SIZE);

  int *d_m, *d_n;
  hipMalloc(&d_m, sizeof(int));
  hipMalloc(&d_n, sizeof(int));
  hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
  d_ActivationFunction<<< gridSize, BLOCK_SIZE >>>(d_A, d_m, d_n);
  hipFree(&d_m);
  hipFree(&d_n);
}

__global__ void d_GetIdentityMatrix(float *d_A, int *d_m, float *d_Alfa){
  int m = (*d_m);
  float alfa = *d_Alfa;
  int bIdx = blockIdx.x;
  int tIdx = threadIdx.x;
  int blockStride = blockDim.x;
  int gridStride = gridDim.x;
  for(int bid=bIdx;bid<m;bid+=gridStride){
    for(int tid=tIdx;tid<m;tid+=blockStride){
      d_A[bid*m + tid] = 0;
      if(bid==tid){
        d_A[bid*m + tid] += alfa;
      }
    }
  }
}

float* getIdentityMatrix(int m, float alfa){
  int gridSize = (m*m/BLOCK_SIZE + 1);
  gridSize = min(gridSize, GRID_SIZE);

  int *d_m;
  hipMalloc(&d_m, sizeof(int));
  hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);

  float *d_Alfa;
  hipMalloc(&d_Alfa, sizeof(float));
  hipMemcpy(d_Alfa, &alfa, sizeof(float), hipMemcpyHostToDevice);

  float *d_A;
  hipMalloc(&d_A, m*m*sizeof(float));
  d_GetIdentityMatrix<<< gridSize, BLOCK_SIZE >>>(d_A, d_m, d_Alfa);
  hipFree(&d_m);
  hipFree(&d_Alfa);
  return d_A;
}

__global__ void d_GetDiagMatrix(float* d_vec, float* diagMat, int *d_m){
  int m = (*d_m);
  int bIdx = blockIdx.x;
  int tIdx = threadIdx.x;
  int blockStride = blockDim.x;
  int gridStride = gridDim.x;
  for(int bid=bIdx;bid<m;bid+=gridStride){
    for(int tid=tIdx;tid<m;tid+=blockStride){
      diagMat[bid*m + tid] = 0;
      if(bid==tid){
        diagMat[bid*m + tid] = d_vec[tid];
      }
    }
  }
}

__global__ void d_GetInverseDiagMatrix(float* d_vec, float* diagMat, int *d_m){
  int m = (*d_m);
  int bIdx = blockIdx.x;
  int tIdx = threadIdx.x;
  int blockStride = blockDim.x;
  int gridStride = gridDim.x;
  for(int bid=bIdx;bid<m;bid+=gridStride){
    for(int tid=tIdx;tid<m;tid+=blockStride){
      diagMat[bid*m + tid] = 0;
      if(bid==tid && abs(d_vec[tid])>EPS){
        diagMat[bid*m + tid] = 1/d_vec[tid];
      }
    }
  }
}

float* diagonalizeVector(float* d_vec, int m, bool inverse){
  /*
    return mxm inverted diagonal matrix made of the input vector
  */
  int gridSize = (m*m/BLOCK_SIZE + 1);
  gridSize = min(gridSize, GRID_SIZE);

  int *d_m;
  hipMalloc(&d_m, sizeof(int));
  hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);

  float* diagMatrix;
  hipMalloc(&diagMatrix, m*m*sizeof(float));

  if (inverse){
    d_GetInverseDiagMatrix<<<gridSize, BLOCK_SIZE>>>(d_vec, diagMatrix, d_m);
  } else {
    d_GetDiagMatrix<<<gridSize, BLOCK_SIZE>>>(d_vec, diagMatrix, d_m);
  }

  hipFree(&d_m);
  return diagMatrix;
}



float* getPseudoInverse(hipblasHandle_t cublasH, hipsolverHandle_t cusolverH,
  float *d_A, int m, int n){

  float *d_At;
  float alfa = 1.0f;
  float beta = 0;
  hipMalloc (&d_At , sizeof(float)*m*n);
  hipblasStatus_t cublasStat = hipblasSgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
    m, n, &alfa, d_A, n, &beta, d_A, m, d_At, m);

  int lwork = 0;
  hipsolverStatus_t cusolverStat = hipsolverDnSgesvd_bufferSize(cusolverH, m, n, &lwork);

  float *d_work, *d_rwork;
  hipMalloc((void**)&d_work , sizeof(float)*lwork);

  float *d_S, *d_U, *d_VT;
  int *devInfo;
  hipMalloc ((void**)&d_S , sizeof(float)*n);
  hipMalloc ((void**)&d_U , sizeof(float)*m*m);
  hipMalloc ((void**)&d_VT , sizeof(float)*m*n);
  hipMalloc ((void**)&devInfo, sizeof(int));

  signed char jobu = 'A'; // all m columns of U
  signed char jobvt = 'A'; // all n columns of VT

  cusolverStat = hipsolverDnSgesvd (
    cusolverH,
    jobu,
    jobvt,
    m,
    n,
    d_At,
    m,
    d_S,
    d_U,
    m, // ldu
    d_VT,
    n, // ldvt,
    d_work,
    lwork,
    d_rwork,
    devInfo);
  printf("%d %d %d\n ", cusolverStat, HIPSOLVER_STATUS_SUCCESS, HIPSOLVER_STATUS_INVALID_VALUE);
  assert(cusolverStat == HIPSOLVER_STATUS_SUCCESS);
  hipFree(&devInfo);
  hipFree(&d_work);
  hipFree(&d_rwork);

  /*
    invert each components
    and then multiply to get pseudo-inverse
  */
  float *d_invDiagS = diagonalizeVector(d_S, n, true);
  hipFree(&d_S);

  float* d_Ainv, *d_Temp;
  hipMalloc(&d_Temp, n*m*sizeof(float));
  hipMalloc(&d_Ainv, n*m*sizeof(float));
  matMul(cublasH, d_invDiagS, d_U, d_Temp, n, n, m, true);
  matMul(cublasH, d_VT, d_Temp, d_Ainv, n, n, m, true);
  hipFree(d_U);
  hipFree(d_VT);
  hipFree(d_Temp);

  return d_Ainv;
}
